#include "hip/hip_runtime.h"
#include "solver.cuh"
#include <math.h>
#include <stdio.h>
using namespace std;
void displayProgressBarGPU(int progress, int total, int barWidth = 60);
__global__ void initia(gpuStatus gpu_data, gpuU gpu_U);
__global__ void GPU_RungeKutta(gpuU gpu_U, int size);

/*constan value stored in GPU*/
extern __constant__ float gpu_R;
extern __constant__ float gpu_gamma;
extern __constant__ float gpu_dx;
extern __constant__ float gpu_dt;

/*update data*/
void Euler::refresh() {
  for (int i = 0; i <= size - 1; i++) {
    data[i].rou = U[i].rou_r;
    data[i].u = U[i].rou_u / U[i].rou_r;
    data[i].E = U[i].rou_e / U[i].rou_r;
    data[i].p =
        (data[i].E * data[i].rou - 0.5 * data[i].rou * pow(data[i].u, 2)) *
        (gamma - 1);
    data[i].c = pow(gamma * data[i].p / data[i].rou, 0.5);
    data[i].T = data[i].p / data[i].rou * gamma * pow(data[i].u / data[i].c, 2);
  }
}

/*construction function*/
Euler::Euler(int size, float t) {
  this->size = size;
  this->t = t;
  dx = 1.0 / size;
  dt = 0.0001;
  data = new status[size];
  U = new Vector[size];
  /*gpu_data*/
  hipMalloc((void **)&(gpu_data.x), size * sizeof(float));
  hipMalloc((void **)&(gpu_data.rou), size * sizeof(float));
  hipMalloc((void **)&(gpu_data.u), size * sizeof(float));
  hipMalloc((void **)&(gpu_data.p), size * sizeof(float));
  hipMalloc((void **)&(gpu_data.E), size * sizeof(float));
  hipMalloc((void **)&(gpu_data.T), size * sizeof(float));
  hipMalloc((void **)&(gpu_data.c), size * sizeof(float));
  /*gpu_u*/
  hipMalloc((void **)&(gpu_U.u1), size * sizeof(float));
  hipMalloc((void **)&(gpu_U.u2), size * sizeof(float));
  hipMalloc((void **)&(gpu_U.u3), size * sizeof(float));
  /*set constant memory*/
  hipMemcpyToSymbol(HIP_SYMBOL(gpu_R), &R0, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(gpu_gamma), &gamma, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(gpu_dx), &dx, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(gpu_dt), &dt, sizeof(float));
  /*event created*/
  hipEventCreate(&kernel_start);
  hipEventCreate(&kernel_end);
  hipEventCreateWithFlags(&kernel_end, hipEventBlockingSync);
  /*get property*/
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
    cerr << "No CUDA-capable devices found." << endl;
    exit(0);
  } else
    cout << deviceCount << " GPUs has been found!" << endl;

  hipGetDeviceProperties(&device_prop, 0);
  cout << "running on " << device_prop.name << endl;
  SM = device_prop.multiProcessorCount;
  cout << "Block number in  grid : Thread in each Block (" << SM << " : "
       << size / SM << ")" << endl;
  /*call gpu_kernel*/
  initia<<<SM, size / SM>>>(gpu_data, gpu_U);
}

/*the process of runge-kutta*/
void Euler::Runge_Kutta() {
  float time;
  hipEventRecord(kernel_start, 0);
  GPU_RungeKutta<<<SM, size / SM>>>(gpu_U, size);
  hipEventRecord(kernel_end, 0);
  hipEventSynchronize(kernel_end);
  hipEventElapsedTime(&time, kernel_start, kernel_end);
  runtime += time;
}
/*solve the equation*/
void Euler::solver() {
  float tt = 0;
  int count = 0;
  while (tt < t) {
    count++;
    displayProgressBarGPU(count, t / dt);
    Runge_Kutta();
    tt = tt + dt;
  }
  cout << endl;
}

/*delete the memory*/
Euler::~Euler() {
  if (data)
    delete[] data;
  if (U)
    delete[] U;
  hipEventDestroy(kernel_start);
  hipEventDestroy(kernel_end);
  hipFree(gpu_data.x);
  hipFree(gpu_data.rou);
  hipFree(gpu_data.u);
  hipFree(gpu_data.p);
  hipFree(gpu_data.E);
  hipFree(gpu_data.T);
  hipFree(gpu_data.c);
  hipFree(gpu_U.u1);
  hipFree(gpu_U.u2);
  hipFree(gpu_U.u3);
}

/*copy data back to cpu*/
void Euler::memCpy() {
  float *tmp = (float *)malloc(size * sizeof(float));
  hipMemcpy(tmp, gpu_data.x, size * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < size; i++)
    data[i].x = tmp[i];
  hipMemcpy(tmp, gpu_U.u1, size * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < size; i++)
    U[i].rou_r = tmp[i];
  hipMemcpy(tmp, gpu_U.u2, size * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < size; i++)
    U[i].rou_u = tmp[i];
  hipMemcpy(tmp, gpu_U.u3, size * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < size; i++)
    U[i].rou_e = tmp[i];
  free(tmp);
  refresh();
}

/*output result*/
void Euler::output(string name) const {
  cout << "runtime on gpu is:" << runtime << "ms" << endl;
  name.append(".dat");
  ofstream out(name);
  for (int i = 0; i <= size - 1; i++)
    // out << data[i].x << "\t" << data[i].p << endl;
    out << data[i].x << '\t' << data[i].u << '\t' << data[i].p << '\t'
        << data[i].rou << '\t' << data[i].T << endl;
  out.close();
}
